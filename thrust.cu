#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

// Enable error checking
#define CUDA_CHECK_ERROR
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 )
    do
    {
        if ( hipSuccess != err )
        {
            fprintf( stderr,
                "cudaSafeCall() failed at %s:%i : %s\n",
                file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );
#pragma warning( pop )
#endif
    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 )
    do
    {
        hipError_t err = hipGetLastError();
        if ( hipSuccess != err )
        {
            fprintf( stderr,
                "cudaCheckError() failed at %s:%i : %s.\n",
                file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
        err = hipDeviceSynchronize();
        if( hipSuccess != err )
        {
            fprintf( stderr,
                "cudaCheckError() with sync failed at %s:%i : %s.\n",
                file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );
#pragma warning( pop )
#endif
    return;
}

int* makeRandArray(const int size, const int seed) {
    srand(seed);
    int* array = new int[size];
    for(int i = 0; i < size; i++) {
        array[i] = std::rand() % 1000000;
    }
    return array;
}

int main(int argc, char* argv[])
{
    int* array;
    int size, seed;

    // Modified argument checking as per PDF requirements
    if(argc < 3) {
        fprintf(stderr, "usage: %s [number of random integers to generate] [seed value for random number generation]\n", 
                argv[0]);
        exit(-1);
    }

    // Parse arguments
    {
        std::stringstream ss1(argv[1]);
        ss1 >> size;
    }
    {
        std::stringstream ss1(argv[2]);
        ss1 >> seed;
    }

    array = makeRandArray(size, seed);

    hipEvent_t startTotal, stopTotal;
    float timeTotal;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventRecord(startTotal, 0);

    // Create device vector and sort
    thrust::device_vector<int> d_vec(array, array + size);
    thrust::sort(d_vec.begin(), d_vec.end());
    
    // Copy back to host array
    thrust::copy(d_vec.begin(), d_vec.end(), array);

    hipEventRecord(stopTotal, 0);
    hipEventSynchronize(stopTotal);
    hipEventElapsedTime(&timeTotal, startTotal, stopTotal);
    hipEventDestroy(startTotal);
    hipEventDestroy(stopTotal);

    fprintf(stderr, "Total time in seconds: %f\n", timeTotal/1000.0);

    // Cleanup
    delete[] array;
    return 0;
}